#include "hip/hip_runtime.h"
//  RunPuzzleParallel.cu
//
//
//  Created by Joey Borowicz on 5/7/17.
//
//
#include <stdio.h>
#include <math.h>
#include <iomanip>
#include <iostream>
#include <string>
#include <sys/time.h>
#include "Puzzles.h"
using namespace std;



// problem size (vector length) N
static const int N = 12345678;


//Note: the first part of this program is essentially copied from the serial version

__device__ bool square(int row, int column, int* puzzle, int counter, int startValue)
{
 if(counter == 81) //went through whole puzzle
    {
        return true;
    }

    //loop of column and rows
    if(++column == 9)
    {
        column = 0;
        if(++row == 9)
        {
            row = 0;
        }
    }

    //skip solved squares
    if(puzzle[column + row * 9] != 0)
    {
        return square(row, column, puzzle, counter+1, startValue);
    }

    for(int i = 1; i <= 9; i++)
    {
        if(++startValue == 10)//This should work to for setting start value at one if cell is 0
        {
            startValue = 1;
        }

        //check if the value is valid using our function
        if(valid(row, column, startValue, puzzle))
        {
            puzzle[column + row * 9] = startValue;

            if(square(row, column, puzzle, counter+1, startValue))
            {
                return true;
            }
        }
    }
    puzzle[column + row * 9] = 0; //set to zero
    //will require backtracking
	
	
	
	
	
	
    return false;
}

__device__ bool valid(int row, int column, int value, int* puzzle)
{
    int i;

    for(i = 0; i < 9; i++)
    {
        if(puzzle[row * 9 + i] == value) //rows
        {
            return false;
        }
        else if(puzzle[column + i * 9] == value) //columns
        {
            return false;
        }
        else if(puzzle[(row/3*3+i%3) * 9 + (column/3*3+i/3) ] == value) //check the subsection 
        {
            return false;
        }
    }
      return true; //valid value
}

//Implementing the parallel solve method
__global__ void solve_parallel(int* puzzle)
{
   	int r = threadIdx.x  //row id
	int c = threadIdx.y  //column id 
 	int s = blockIdx.x * blockDum.x + threadIdx.x  //setting the start value
		
	if(square(r,c,puzzle,0, s)) 
     	{
        	cout << "Puzzle Solved\n";
    	}
    	else 
    	{
       	 	cout << "Puzzle Not Solved\n";
    	}
		
}



/*__global__ bool valid_parallel(int *puzzle,int value, int *output)
{
	int r = threadIdx.x  //row id
	int c = threadIdx.y  //column id 
	int s = blockIdx.x * blockDum.x + threadIdx.x  //setting the start value using the idea from assignment1
	

	
	if(puzzle[r * 9 + s] == value) //rows
        {
            	return false;
        }
        else if(puzzle[c + s * 9] == value) //columns
        {
            	return false;
        }
        else if(puzzle[(r/3*3+s%3) * 9 + (c/3*3+s/3) ] == value) //check the subsection 
        {
            	return false;
        }
	return true;

}*/


__device__ void display(int* puzzle)
{
for (int h = 0; h < 81; h++)
{
 if (h % 27 == 0)
	{
		cout << "\n-------------------------";
  	}
 if (h % 9 == 0)
  	{
  		cout << "\n";
		cout << "| "; 
	}
 cout << puzzle[h];
 cout << " "; 
 if (h % 3 == 2)
	{
  		cout << "| ";
	}
}
cout << "\n";
cout << "-------------------------";
cout << "\n";
}


//this was used in both of the 2 CUDA assignments
// Returns the current time in microseconds
long long start_timer() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec * 1000000 + tv.tv_usec;
}


// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, std::string name) {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
    std::cout << std::setprecision(5) << std::fixed;
	std::cout << name << ": " << ((float) (end_time - start_time)) / (1000) << " Msec\n";
	return end_time - start_time;
}


//TODO: Memory allocation and the other cuda specific memory operations. Need to look closer at some other resources.

int main()
{
	
	//CPU Implementation
	Puzzles p;
	int* h_puzzle = (int*)malloc(81*sizeof(int));
	
	//Initializing data on CPU
	int i;
	for(i = 0; i < 81; i++)
	{
		h_puzzle[i] = p.puzzleOne[i];
	}
	
	//Execute and time: CPU version
	std::clock_t CPU_start;
	double CPU_totalTime;
	CPU_start = clock();
	solve(h_puzzle);
	display(h_puzzle);
	CPU_totalTime = (clock() - CPU_start) / (double) CLOCKS_PER_SEC;
	cout << "\nTime: " << CPU_totalTime << " seconds\n";
	
	
	//GPU Implementation
	
	//not sure if this is required
	const int sizeOfBlock = 1024;
  	const int sizeOfGrid = N/1024 + 1; 
  	const float bytes = 81*sizeof(int);
	long long GPU_startTotal = start_timer(); //GPU start time
	
	//Allocating memory to GPU and timing it
	long long GPU_allocateStart = start_timer();
	int* d_puzzle = (int*)malloc(81*sizeof(int)); 
	hipMalloc((void**) &d_input, bytes); 
	long long GPU_allocateTime = stop_timer(GPU_allocateStart, "\nGPU Memory Allocation");
	
	
	long long GPU_dcopyStart = start_timer();
	hipMemcpy(d_puzzle, h_puzzle, 81*sizeof(int), hipMemcpyHostToDevice);
	long long GPU_dcopyTime = stop_timer(GPU_dcopyStart, "Copying GPU Memory to Device"); 
	
	
	long long GPU_kernelStart = start_timer();
	solve_parallel<<<sizeOfGrid, sizeOfBlock>>>(d_puzzle);
	display<<<sizeOfGrid, sizeOfBlock>>>(h_puzzle);
	long long GPU_kernelTime = stop_timer(GPU_kernelStart, "GPU Kernel Run Time");
	
	
	//Copying the output to the host
  	long long GPU_hcopyStart = start_timer();
  	hipMemcpy(h_gpu_result, d_output, bytes, hipMemcpyDeviceToHost);
  	long long GPU_hcopyTime = stop_timer(GPU_hcopyStart, "Copying GPU Memory to Host");
	
	//Free GPU memory
 	hipFree(d_puzzle);
	
	// End GPU timer
  	long long GPU_totalTime = stop_timer(GPU_startTotal, "Total GPU Run Time");
	
	/*
	// Checking to make sure the CPU and GPU results match - Do not modify
  	int errorCount = 0;
  	for (i=0; i<N; i++)
  	{
   		 if (abs(h_cpu_result[i]-h_gpu_result[i]) > 1e-6)
      		 errorCount = errorCount + 1;
  	}
  	if (errorCount > 0)
   	printf("Result comparison failed.\n");
  	else
    	printf("Result comparison passed.\n");
	*/
	
	
	 // Cleaning up memory
  	free(h_puzzle);
  	//free(h_cpu_result);
  	//free(h_gpu_result);
  	return 0;
	
	
	/*//calculating time taken 
	std::clock_t GPU_start;
	double GPU_totalTime;
	GPU_start = clock();
	parallel_solve(puzzle);
	display(puzzle);
	GPU_totalTime = (clock() - GPU_start) / (double) CLOCKS_PER_SEC;
	cout << "\nTime: " << GPU_totalTime << " seconds\n";*/
	

  	
	
}














