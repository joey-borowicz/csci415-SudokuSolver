#include "hip/hip_runtime.h"
//  RunPuzzleParallel.cu
//
//
//  Created by Joey Borowicz on 5/7/17.
//
//
#include <stdio.h>
#include <math.h>
#include <iomanip>
#include <iostream>
#include <string>
#include <sys/time.h>
#include "Puzzles.h"
using namespace std;

//Note: the first part of this program is essentially copied from the serial version

__device__ bool square(int row, int column, int* puzzle, int counter, int startValue)
{
 if(counter == 81) //went through whole puzzle
    {
        return true;
    }

    //loop of column and rows
    if(++column == 9)
    {
        column = 0;
        if(++row == 9)
        {
            row = 0;
        }
    }

    //skip solved squares
    if(puzzle[column + row * 9] != 0)
    {
        return square(row, column, puzzle, counter+1, startValue);
    }

    for(int i = 1; i <= 9; i++)
    {
        if(++startValue == 10)//This should work to for setting start value at one if cell is 0
        {
            startValue = 1;
        }

        //check if the value is valid using our function
        if(valid(row, column, startValue, puzzle))
        {
            puzzle[column + row * 9] = startValue;

            if(square(row, column, puzzle, counter+1, startValue))
            {
                return true;
            }
        }
    }
    puzzle[column + row * 9] = 0; //set to zero
    //will require backtracking
    return false;
}

__device__ bool valid(int row, int column, int value, int* puzzle)
{
    int i;

    for(i = 0; i < 9; i++)
    {
        if(puzzle[row * 9 + i] == value) //rows
        {
            return false;
        }
        else if(puzzle[column + i * 9] == value) //columns
        {
            return false;
        }
        else if(puzzle[(row/3*3+i%3) * 9 + (column/3*3+i/3) ] == value) //check the subsection 
        {
            return false;
        }
    }
      return true; //valid value
}

__global__ bool sudoku_parallel(int *puzzle, int *output)
{
	int r = threadIdx.x  //row id
	int c = threadIdx.y  //column id 
	int s = blockIdx.x * blockDum.x + threadIdx.x  //setting the start value using the idea from assignment1
	

	//Implementing the parallel code
		if(puzzle[row * 9 + s] == value) //rows
        	{
            		return false;
        	}
        	else if(puzzle[column + s * 9] == value) //columns
        	{
            		return false;
        	}
        	else if(puzzle[(row/3*3+s%3) * 9 + (column/3*3+s/3) ] == value) //check the subsection 
        	{
            		return false;
        	}
		return true;

}


__device__ void display(int* puzzle)
{
for (int h = 0; h < 81; h++)
{
 if (h % 27 == 0)
	{
		cout << "\n-------------------------";
  	}
 if (h % 9 == 0)
  	{
  		cout << "\n";
		cout << "| "; 
	}
 cout << puzzle[h];
 cout << " "; 
 if (h % 3 == 2)
	{
  		cout << "| ";
	}
}
cout << "\n";
cout << "-------------------------";
cout << "\n";
}


//this was used in both of the 2 CUDA assignments
// Returns the current time in microseconds
long long start_timer() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec * 1000000 + tv.tv_usec;
}


// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, std::string name) {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
    std::cout << std::setprecision(5) << std::fixed;
	std::cout << name << ": " << ((float) (end_time - start_time)) / (1000) << " Msec\n";
	return end_time - start_time;
}


//TODO: Memory allocation and the other cuda specific memory operations. Need to look closer at some other resources.
