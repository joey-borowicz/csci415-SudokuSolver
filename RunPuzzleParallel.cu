//  RunPuzzleParallel.cu
//
//
//  Created by Joey Borowicz on 5/7/17.
//
//

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <iomanip>
#include <iostream>
#include <string>
#include <sys/time.h>
#include <unistd.h>
#include <sys/types.h>
#include <cstdio>
#include <iostream>

using namespace std;

//Note: the first part of this program is essentially copied from the serial version

//I moved this to the top because of an error I was getting
__device__ bool valid(int row, int column, int value, int* puzzle)
{
    for(int i = 0; i < 9; i++)
    {
        if(puzzle[row * 9 + i] == value) //rows
        {
            return false;
        }
        else if(puzzle[column + i * 9] == value) //columns
        {
            return false;
        }
        else if(puzzle[(row/3*3+i%3) * 9 + (column/3*3+i/3) ] == value) //check the subs$
        {
            return false;
        }
    }
      return true; //valid value
}

__device__ bool square(int row, int column, int* puzzle, int counter, int startValue)
{
 if(counter == 81) //went through whole puzzle
    {
        return true;
    }

    //loop of column and rows
    if(++column == 9)
    {
        column = 0;
        if(++row == 9)
        {
            row = 0;
        }
    }

    //skip solved squares
    if(puzzle[column + row * 9] != 0)
    {
        return square(row, column, puzzle, counter+1, startValue);
    }

    for(int i = 1; i <= 9; i++)
    {
        if(++startValue == 10)//This should work to for setting start value at one if cell is 0
        {
            startValue = 1;
        }

        //check if the value is valid using our function
        if(valid(row, column, startValue, puzzle))
        {
            puzzle[column + row * 9] = startValue;

            if(square(row, column, puzzle, counter+1, startValue))
            {
                return true;
            }
        }
    }
    puzzle[column + row * 9] = 0; //set to zero
    //will require backtracking
	
	
	
	
	
	
    return false;
}


//Implementing the parallel solve method
__global__ void solve_parallel(int* puzzle, int* output)
{
   	int r = threadIdx.x;  //row id
	int c = threadIdx.y;  //column id 
 	int s = (blockIdx.x * blockDim.x + threadIdx.x) % 9 + 1;  //setting the start value
	int resultIndicator;

	if(square(r,c,puzzle,0, s)) 
     	{

        	resultIndicator = 1;
		(*output) = resultIndicator;
    	}
    	else 
    	{
       	 	resultIndicator = 0;
		(*output) = resultIndicator;
    	}
		
}



/*__global__ bool valid_parallel(int *puzzle,int value, int *output)
{
	int r = threadIdx.x  //row id
	int c = threadIdx.y  //column id 
	int s = blockIdx.x * blockDum.x + threadIdx.x  //setting the start value using the idea from assignment1
	

	
	if(puzzle[r * 9 + s] == value) //rows
        {
            	return false;
        }
        else if(puzzle[c + s * 9] == value) //columns
        {
            	return false;
        }
        else if(puzzle[(r/3*3+s%3) * 9 + (c/3*3+s/3) ] == value) //check the subsection 
        {
            	return false;
        }
	return true;

}*/


void display(int* puzzle)
{
for (int h = 0; h < 81; h++)
{
 if (h % 27 == 0)
	{
		cout << "\n-------------------------";
  	}
 if (h % 9 == 0)
  	{
  		cout << "\n";
		cout << "| "; 
	}
 cout << puzzle[h];
 cout << " "; 
 if (h % 3 == 2)
	{
  		cout << "| ";
	}
}
cout << "\n";
cout << "-------------------------";
cout << "\n";
}


//this was used in both of the 2 CUDA assignments
long long start_timer() {
        struct timeval tv;
        gettimeofday(&tv, NULL);
        return tv.tv_sec * 1000000 + tv.tv_usec;
}


// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, std::string name) {
        struct timeval tv;
        gettimeofday(&tv, NULL);
        long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
        std::cout << std::setprecision(5);
        std::cout << name << ": " << ((float) (end_time - start_time)) / (1000 * 1000) << " sec\n";
        return end_time - start_time;
}

//TODO: Memory allocation and the other cuda specific memory operations. Need to look closer at some other resources.

int main()
{
	
	//CPU Implementation
	int original[81] = {0,2,0,6,0,8,0,0,0,
                     5,8,0,0,0,9,7,0,0,
                     0,0,0,0,4,0,0,0,0,
                     3,7,0,0,0,0,5,0,0,
                     6,0,0,0,0,0,0,0,4,
                     0,0,8,0,0,0,0,1,3,
                     0,0,0,0,2,0,0,0,0,
                     0,0,9,8,0,0,0,3,6,
                     0,0,0,3,0,6,0,9,0}; //taken from Puzzles.h since that format won't work here
	
	int originalTwo[81] = {1,0,0,4,0,0,2,0,9,
				0,0,7,0,0,0,0,0,0,
				5,8,9,0,0,0,1,0,0,
				0,0,0,0,0,0,3,9,0,
				7,0,0,0,0,1,5,0,0,
				0,4,0,6,0,0,0,0,2,
				9,6,0,0,5,0,0,0,0,
				0,0,5,0,0,8,0,0,0,
				3,7,0,0,2,0,9,6,0};	
	int* puzzle = (int*)malloc(81*sizeof(int));
	
	//Initializing data on CPU
	int i;
	for(i = 0; i < 81; i++)
	{
		puzzle[i] = originalTwo[i];
	}
	
	//int* test = (int*)malloc(81*sizeof(int));
	//test = puzzle;
	
	//GPU implementation

        int* h_puzzle = (int*)malloc(81*sizeof(int)); //h is for host vars
	int* h_output = (int*)malloc(sizeof(int)); 
	int* m_output; //m vars will deal with memory
	int* m_puzzle;
        int test_output = 0;
        h_output = &test_output;

	long long GPU_total_start = start_timer(); //taken from what I did in assignment1
	
	hipMalloc((void**) &m_puzzle, 81*sizeof(int));
	hipMalloc((void**) &m_output, sizeof(int));
	
	hipMemcpy(m_puzzle, puzzle, 81*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(m_output, h_output, sizeof(int), hipMemcpyHostToDevice);
	
        // warning : Stack size for entry function '_Z14solve_parallelPiS_' cannot be statically $
        size_t stack = 12345;
        hipDeviceSetLimit(hipLimitStackSize, stack);
	solve_parallel<<<1, 9>>>(m_puzzle, m_output); //1block, 9threads
	hipDeviceSynchronize();


	hipMemcpy(h_puzzle, m_puzzle, 81*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_output, m_output, sizeof(int), hipMemcpyDeviceToHost);

	long long GPU_total_time = stop_timer(GPU_total_start, "\nTotal time");

	if(*h_output == 1)
	{
		cout << "Puzzle Solved\n";
	}
	else
	{
		cout << "Puzzle Not Solved\n";
	}
	
	display(h_puzzle);
/*
	free(h_puzzle);
	free(h_output);
	free(m_puzzle);
	free(m_output);
	cudaFree(h_puzzle);
	cudaFree(h_output);
	cudaFree(m_puzzle);
	cudaFree(m_output);
*/
	return 0;
}
/*
	//not sure if this is required
	const int sizeOfBlock = 1024;
  	const int sizeOfGrid = N/1024 + 1;
  	const float bytes = 81*sizeof(int);
	long long GPU_startTotal = start_timer(); //GPU start time
	
	//Allocating memory to GPU and timing it
	long long GPU_allocateStart = start_timer();
	int* d_puzzle = (int*)malloc(81*sizeof(int)); 
	cudaMalloc((void**) &d_input, bytes); 
	long long GPU_allocateTime = stop_timer(GPU_allocateStart, "\nGPU Memory Allocation");
	
	
	long long GPU_dcopyStart = start_timer();
	cudaMemcpy(d_puzzle, h_puzzle, 81*sizeof(int), cudaMemcpyHostToDevice);
	long long GPU_dcopyTime = stop_timer(GPU_dcopyStart, "Copying GPU Memory to Device"); 
	
	
	long long GPU_kernelStart = start_timer();
	solve_parallel<<<sizeOfGrid, sizeOfBlock>>>(d_puzzle);
	display<<<sizeOfGrid, sizeOfBlock>>>(h_puzzle);
	long long GPU_kernelTime = stop_timer(GPU_kernelStart, "GPU Kernel Run Time");
	
	
	//Copying the output to the host
  	long long GPU_hcopyStart = start_timer();
  	cudaMemcpy(h_gpu_result, d_output, bytes, cudaMemcpyDeviceToHost);
  	long long GPU_hcopyTime = stop_timer(GPU_hcopyStart, "Copying GPU Memory to Host");
	
	//Free GPU memory
 	cudaFree(d_puzzle);
	
	// End GPU timer
  	long long GPU_totalTime = stop_timer(GPU_startTotal, "Total GPU Run Time");
*/	
	/*
	// Checking to make sure the CPU and GPU results match - Do not modify
  	int errorCount = 0;
  	for (i=0; i<N; i++)
  	{
   		 if (abs(h_cpu_result[i]-h_gpu_result[i]) > 1e-6)
      		 errorCount = errorCount + 1;
  	}
  	if (errorCount > 0)
   	printf("Result comparison failed.\n");
  	else
    	printf("Result comparison passed.\n");
	*/

	
	
	/*//calculating time taken 
	std::clock_t GPU_start;
	double GPU_totalTime;
	GPU_start = clock();
	parallel_solve(puzzle);
	display(puzzle);
	GPU_totalTime = (clock() - GPU_start) / (double) CLOCKS_PER_SEC;
	cout << "\nTime: " << GPU_totalTime << " seconds\n";*/	
